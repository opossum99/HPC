#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "glad/glad.h"
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>
#include "solver.cuh"
#include "gl_viewer.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template<typename T>
__global__ void init_grad_pos(T * array, T h, size_t N, size_t M)
{
  int numThreads = blockDim.x * gridDim.x;
  int global_id = threadIdx.x + blockIdx.x * blockDim.x;

  for (int id = global_id; id < N*M; id+=numThreads)
  {
    int i = id/N;
    int j = id%N;
    array[3*(j+i*N) + 0] = j*h;
    array[3*(j+i*N) + 1] = i*h;
    array[3*(j+i*N) + 2] = 0.0;
  }
  return;
}

template<typename T, typename S>
__global__ void update_grad_dir(T * array, T h, size_t N, size_t M, S * ey)
{
  int numThreads = blockDim.x * gridDim.x;
  int global_id = threadIdx.x + blockIdx.x * blockDim.x;

  for (int id = global_id; id < (N-2)*(M-2); id+=numThreads)
  {
    int i = id/(N-2)+1;
    int j = id%(N-2)+1;
    float scale = 0.0005;
    array[3*(j+i*N) + 0 + N*M*3] = array[3*(j+i*N) + 0] + scale*(ey[j+1+i*N]-ey[j-1+i*N])/(2.*h);
    array[3*(j+i*N) + 1 + N*M*3] = array[3*(j+i*N) + 1] + scale*(ey[j+(i+1)*N]-ey[j+(i+1)*N])/(2.*h);
    //array[3*(j+i*N) + 2 + N*M*3] = 0.0;
  }
  return;
}



int main(int argc, char ** argv)
{
  std::cout << "inside main\n";

  custom_gl::gl_viewer viewer;//(true);
  int N = 200*10;
  int M = 100*10;
  float h = 0.01/10;
  float dt = 0.004/10;
  int TIME = 1000;

  gpuErrchk(hipSetDevice(0));
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("Device Number: %d\n", 0);
  printf("  Device name: %s\n", prop.name);
  printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
  printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
  printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 
      2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);


  solver::gpu_solver<float> solver(N, M, h, dt);

  dim3 block(256);
  dim3 grid((N*M + block.x-1)/block.x);

  unsigned int * ebo = new unsigned int [N*M*2];
  for (int i = 0; i < N*M; i++)
  {
    ebo[2*i] = i;
    ebo[2*i+1] = i+N*M;
  }

  float * grads_gl;
  hipGraphicsResource *vbo_res;  
  viewer.buffer_vbo(N*M*2*3);

  gpuErrchk(hipGraphicsGLRegisterBuffer(&vbo_res, viewer.VBO, cudaGraphicsMapFlagsNone));
  gpuErrchk(hipGraphicsMapResources(1, &vbo_res, 0));
  size_t num_bytes;
  gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&grads_gl, &num_bytes, vbo_res));

  viewer.buffer_ebo(N*M*2, ebo);
  
  init_grad_pos<<<grid, block>>>(grads_gl, h, N, M);
  init_grad_pos<<<grid, block>>>(&grads_gl[N*M*3], h, N, M);

  viewer.preset();
  
  //while (!glfwWindowShouldClose(viewer.window))
  //{
    hipDeviceSynchronize();
    clock_t start = clock();
    for (int t = 0; t < TIME; t++)
    {
      dim3 grid_i( ((N-2)*(M-2) + block.x-1)/block.x);
      bool to_save = false;//t % 10 == 0;//false;//
      solver.iteration(t, to_save);
      update_grad_dir<<<grid_i, block>>>(grads_gl, h, N, M, solver.ey);
      hipDeviceSynchronize();
      viewer.view(N*M*2*3);
    }
    hipDeviceSynchronize();
    std::cout << "Total computation time: " << (double) (clock() - start)/CLOCKS_PER_SEC << "\n";
  //}
  gpuErrchk(hipGraphicsUnmapResources(1, &vbo_res, 0));
  delete [] ebo;
  return 0;
}
